#include "hip/hip_runtime.h"
/*
 * MultiNBodyDomain.cpp
 *
 *  Created on: Oct 15, 2012
 *      Author: martin
 */

#include "MultiNBodyDomain.h"
#include "MultiNBodyWorld.h"

#ifdef USE_SDK
#include <hip/hip_runtime_api.h>
#endif

MultiNBodyDomain::MultiNBodyDomain():Domain(),np(0)
{
	// TODO Auto-generated constructor stub
}

MultiNBodyDomain::~MultiNBodyDomain()
{
	// TODO Auto-generated destructor stub
	if(isActive()){
		if(h_pos){delete h_pos;h_pos=NULL;}
		if(h_mom){delete h_mom;h_mom=NULL;}
		if(h_force){delete h_force;h_force=NULL;}
		if(onGpu()){
			if(d_pos[0]){hipFree(d_pos[0]);d_pos[0]=NULL;}
			if(d_pos[1]){hipFree(d_pos[1]);d_pos[1]=NULL;}
			if(d_mom){hipFree(d_mom);d_mom=NULL;}
			if(d_force){hipFree(d_force);d_force=NULL;}
		}
	}
}

void MultiNBodyDomain::activate(){
	if(rank==theWorld->getRank())activated=true;
	if(isActive()){  // Only Allocate Memory on Activated Domains on Process
		//printf("Allocating CPU Data\n");
		h_pos=new float[4*np]; // x,y,z,mass
		h_mom=new float[4*np]; // px,py,pz,energy
		h_force=new float[4*np]; // fx,fy,fz,n/a - 4 not 4 for memory alignment

		if(onGpu()){
			//printf("Allocating GPU Data\n");

			int count;
			hipGetDeviceCount(&count);
			dev=rank%count;
			hipDeviceProp_t props;
			hipGetDeviceProperties(&props, dev);
			// Debug - Display Compute level by card
			// Compute 1.3 incompatible with kernel printf
			//printf("Device Properties[%d]: Compute:%d.%d\n",dev,props.major,props.minor);

			hipSetDevice(dev);

			hipMalloc(&d_pos[0],4*np*sizeof(float));
			hipMalloc(&d_pos[1],4*np*sizeof(float));
			hipMalloc(&d_mom,4*np*sizeof(float));
			hipMalloc(&d_force,4*np*sizeof(float));
		}
	}
}

void MultiNBodyDomain::init(){
	int setcfg=theWorld->getRank()%NBODY_NUM_CONFIGS;
	if(isActive())randomizeBodies(setcfg);
}

void MultiNBodyDomain::step(float indt,int di){
	if(isActive()){
		if(onGpu()) gpuIntegrateNBodySystem(indt,di);
		else integrateNBodySystem(indt, di);

		// To use force calculation only on GPU simply call:
		//integrateNBodySystem(indt);
		// Note: Thread block sizes not updated for arbitrary np in force calculator kernel call -- use power of 2

// ----DEBUG--- Print a particle position per domain for debugging
		//printf("{%f %f %f} ",h_pos[4*(np-1)],h_pos[4*(np-1)+1],h_pos[4*(np-1)+2]);
	}
}


// Much of the following is a stripped down version of the CUDA SDK NBody Demo

// Macros to simplify shared memory addressing
#define SX(i) sharedPos[i+blockDim.x*threadIdx.y]
// This macro is only used when multithreadBodies is true (below)
#define SX_SUM(i,j) sharedPos[i+blockDim.x*j]

struct SharedMemory
{
    __device__ inline operator       float4 *()
    {
        extern __shared__ int __smem[];
        return (float4 *)__smem;
    }

    __device__ inline operator const float4 *() const
    {
        extern __shared__ int __smem[];
        return (float4 *)__smem;
    }
};

__device__ float4 bodyBodyInteraction(float4 ai,float4 bi,float4 bj,float softeningSquared)
{
    float4 r;

    // r_ij  [3 FLOPS]
    r.x = bj.x - bi.x;
    r.y = bj.y - bi.y;
    r.z = bj.z - bi.z;

    // distSqr = dot(r_ij, r_ij) + EPS^2  [6 FLOPS]
    float distSqr = r.x * r.x + r.y * r.y + r.z * r.z;
    distSqr += softeningSquared;

    // invDistCube =1/distSqr^(3/2)  [4 FLOPS (2 mul, 1 sqrt, 1 inv)]
    float invDist = rsqrt(distSqr);
    float invDistCube =  invDist * invDist * invDist;

    // s = m_j * invDistCube [1 FLOP]
    float s = bj.w * invDistCube;

    // a_i =  a_i + s * r_ij [6 FLOPS]
    ai.x += r.x * s;
    ai.y += r.y * s;
    ai.z += r.z * s;

    return ai;
}

// This is the "tile_calculation" function from the GPUG3 article.
__device__ float4 gravitation(float4 iPos,float4 accel,float ssq)
{
    float4 *sharedPos = SharedMemory();

    // The CUDA 1.1 compiler cannot determine that i is not going to
    // overflow in the loop below.  Therefore if int is used on 64-bit linux
    // or windows (or long instead of long long on win64), the compiler
    // generates suboptimal code.  Therefore we use long long on win64 and
    // long on everything else. (Workaround for Bug ID 347697)
#ifdef _Win64
    unsigned long long j = 0;
#else
    unsigned long j = 0;
#endif

    // Here we unroll the loop to reduce bookkeeping instruction overhead
    // 32x unrolling seems to provide best performance

    // Note that having an unsigned int loop counter and an unsigned
    // long index helps the compiler generate efficient code on 64-bit
    // OSes.  The compiler can't assume the 64-bit index won't overflow
    // so it incurs extra integer operations.  This is a standard issue
    // in porting 32-bit code to 64-bit OSes.

#pragma unroll 32

    for (unsigned int counter = 0; counter < blockDim.x; counter++)
    {
        accel = bodyBodyInteraction(accel, iPos, SX(j++),ssq);
    }

    return accel;
}

// WRAP is used to force each block to start working on a different
// chunk (and wrap around back to the beginning of the array) so that
// not all multiprocessors try to read the same memory locations at
// once.
#define WRAP(x,m) (((x)<m)?(x):(x-m))  // Mod without divide, works on values from 0 up to 2m

__device__ float4 computeBodyAccel(float4 bodyPos,
                 float4 *positions,
                 int numBodies,float ssq)
{
    float4 *sharedPos = SharedMemory();

    float4 acc = {0.0f, 0.0f, 0.0f, 0.0f};

    int p = blockDim.x;
    int q = blockDim.y;
    int n = numBodies;
    int numTiles = n / (p * q);

    for (int tile = blockIdx.y; tile < numTiles + blockIdx.y; tile++)
    {
        sharedPos[threadIdx.x+blockDim.x*threadIdx.y] =
            positions[WRAP(blockIdx.x + q * tile + threadIdx.y, gridDim.x) * p + threadIdx.x] ;

        __syncthreads();

        // This is the "tile_calculation" function from the GPUG3 article.
        acc = gravitation(bodyPos, acc, ssq);

        __syncthreads();
    }

    // When the numBodies / thread block size is < # multiprocessors (16 on G80), the GPU is
    // underutilized.  For example, with a 256 threads per block and 1024 bodies, there will only
    // be 4 thread blocks, so the GPU will only be 25% utilized. To improve this, we use multiple
    // threads per body.  We still can use blocks of 256 threads, but they are arranged in q rows
    // of p threads each.  Each thread processes 1/q of the forces that affect each body, and then
    // 1/q of the threads (those with threadIdx.y==0) add up the partial sums from the other
    // threads for that body.  To enable this, use the "--p=" and "--q=" command line options to
    // this example. e.g.: "nbody.exe --n=1024 --p=64 --q=4" will use 4 threads per body and 256
    // threads per block. There will be n/p = 16 blocks, so a G80 GPU will be 100% utilized.

    // We use a bool template parameter to specify when the number of threads per body is greater
    // than one, so that when it is not we don't have to execute the more complex code required!

	SX_SUM(threadIdx.x, threadIdx.y).x = acc.x;
	SX_SUM(threadIdx.x, threadIdx.y).y = acc.y;
	SX_SUM(threadIdx.x, threadIdx.y).z = acc.z;

	__syncthreads();

	// Save the result in global memory for the integration step
	if (threadIdx.y == 0)
	{
		for (int i = 1; i < blockDim.y; i++)
		{
			acc.x += SX_SUM(threadIdx.x,i).x;
			acc.y += SX_SUM(threadIdx.x,i).y;
			acc.z += SX_SUM(threadIdx.x,i).z;
		}
	}

    return acc;
}

__global__ void calculateForce(float4 *pos,
                float4 *force,
                unsigned int deviceNumBodies, int totalNumBodies, float ssq)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    //printf("Index (%d)\n",index);  // Debug that Kernel did not crash if not using SDK
    if (index >= totalNumBodies)
    {
        return;
    }

    float4 position = pos[index];

    force[index]= computeBodyAccel(position, pos, totalNumBodies, ssq);
}

void MultiNBodyDomain::gpuComputeNBodyGravitation(){
	hipMemcpy(d_pos[0],h_pos,4*np*sizeof(float),hipMemcpyHostToDevice);

// Port Force Calculation Only
	hipDeviceProp_t props;

	//unsigned int dev = 0;
	hipGetDeviceProperties(&props, dev);

	int q=props.multiProcessorCount; // Use all multiprocsessors
	int p=min(props.maxThreadsDim[0],(int)ceil((float)np/(float)q));

	int pq=min(props.maxThreadsPerBlock,p*q);
	p=pq/q;

	//printf("Blocking: p=%d,q=%d,p*q=%d - np=%d\n",p,q,p*q,np);

	dim3 threads(p,q,1);
	dim3 grid((np + (p-1))/p, 1, 1);


	// execute the kernel:

	// When the numBodies / thread block size is < # multiprocessors
	// (16 on G80), the GPU is underutilized. For example, with 256 threads per
	// block and 1024 bodies, there will only be 4 thread blocks, so the
	// GPU will only be 25% utilized.  To improve this, we use multiple threads
	// per body.  We still can use blocks of 256 threads, but they are arranged
	// in q rows of p threads each.  Each thread processes 1/q of the forces
	// that affect each body, and then 1/q of the threads (those with
	// threadIdx.y==0) add up the partial sums from the other threads for that
	// body.  To enable this, use the "--p=" and "--q=" command line options to
	// this example.  e.g.: "nbody.exe --n=1024 --p=64 --q=4" will use 4
	// threads per body and 256 threads per block. There will be n/p = 16
	// blocks, so a G80 GPU will be 100% utilized.

	// We use a bool template parameter to specify when the number of threads
	// per body is greater than one, so that when it is not we don't have to
	// execute the more complex code required!
	int sharedMemSize = p * q * 4 * sizeof(float); // 4 floats for pos

	if (grid.x > 0)
	{
		calculateForce<<< grid, threads, sharedMemSize >>>((float4 *)d_pos[0], (float4 *)d_force,p,np,param.m_softeningSquared);
	}

#ifdef USE_SDK
	getLastCudaError("Kernel execution failed");
#endif

	hipMemcpy(h_force,d_force,(4*np)*sizeof(float),hipMemcpyDeviceToHost);
}

void MultiNBodyDomain::computeNBodyGravitation()
{
#ifdef OPENMP
    #pragma omp parallel for
#endif

    for (int i = 0; i < np; i++)
    {
        int indexForce = 4*i;

        float acc[3] = {0, 0, 0};

        // We unroll this loop 4X for a small performance boost.
        int j = 0;

        while (j < np)
        {
            bodyBodyInteraction(acc, &h_pos[4*i], &h_pos[4*j], param.m_softeningSquared);
            j++;
            bodyBodyInteraction(acc, &h_pos[4*i], &h_pos[4*j], param.m_softeningSquared);
            j++;
            bodyBodyInteraction(acc, &h_pos[4*i], &h_pos[4*j], param.m_softeningSquared);
            j++;
            bodyBodyInteraction(acc, &h_pos[4*i], &h_pos[4*j], param.m_softeningSquared);
            j++;
        }

        h_force[indexForce  ] = acc[0];
        h_force[indexForce+1] = acc[1];
        h_force[indexForce+2] = acc[2];
    }
}

void MultiNBodyDomain::bodyBodyInteraction(float accel[3], float posMass0[4], float posMass1[4], float softeningSquared)
{
    float r[3];

    // r_01  [3 FLOPS]
    r[0] = posMass1[0] - posMass0[0];
    r[1] = posMass1[1] - posMass0[1];
    r[2] = posMass1[2] - posMass0[2];

    // d^2 + e^2 [6 FLOPS]
    float distSqr = r[0] * r[0] + r[1] * r[1] + r[2] * r[2];
    distSqr += softeningSquared;

    // invDistCube =1/distSqr^(3/2)  [4 FLOPS (2 mul, 1 sqrt, 1 inv)]
    float invDist = (float)1.0 / (float)sqrt((double)distSqr);
    float invDistCube =  invDist * invDist * invDist;

    // s = m_j * invDistCube [1 FLOP]
    float s = posMass1[3] * invDistCube;

    // (m_1 * r_01) / (d^2 + e^2)^(3/2)  [6 FLOPS]
    accel[0] += r[0] * s;
    accel[1] += r[1] * s;
    accel[2] += r[2] * s;
}

void MultiNBodyDomain::integrateNBodySystem(float deltaTime,int di)
{
    if(onGpu()) gpuComputeNBodyGravitation();
    else computeNBodyGravitation();

    // Debug Force should match GPU vs CPU
    //printf("Force: %f %f %f [%d]\n",h_force[0+4*(np-1)],h_force[1+4*(np-1)],h_force[2+4*(np-1)],np-1);

#ifdef OPENMP
    #pragma omp parallel for
#endif

    for (int i = 0; i < np; ++i)
    {
        int index = 4*i;
        int indexForce = 3*i;


        float pos[3], vel[3], force[3];
        pos[0] = h_pos[index+0];
        pos[1] = h_pos[index+1];
        pos[2] = h_pos[index+2];
        float invMass = 1.0f/h_pos[index+3];

        vel[0] = h_mom[index+0];
        vel[1] = h_mom[index+1];
        vel[2] = h_mom[index+2];

        force[0] = h_force[index+0];//h_force[indexForce+0];
        force[1] = h_force[index+1];//h_force[indexForce+1];
        force[2] = h_force[index+2];//h_force[indexForce+2];

        // acceleration = force / mass;
        // new velocity = old velocity + acceleration * deltaTime
        vel[0] += (force[0] * invMass) * deltaTime;
        vel[1] += (force[1] * invMass) * deltaTime;
        vel[2] += (force[2] * invMass) * deltaTime;

        vel[0] *= param.m_damping;
        vel[1] *= param.m_damping;
        vel[2] *= param.m_damping;

        // new position = old position + velocity * deltaTime
        pos[0] += vel[0] * deltaTime;
        pos[1] += vel[1] * deltaTime;
        pos[2] += vel[2] * deltaTime;

        h_pos[index+0] = pos[0];
        h_pos[index+1] = pos[1];
        h_pos[index+2] = pos[2];

        h_mom[index+0] = vel[0]*h_pos[index+3];
        h_mom[index+1] = vel[1]*h_pos[index+3];
        h_mom[index+2] = vel[2]*h_pos[index+3];
    }
    //printf("Point 0: %f,%f,%f\n",h_pos[0],h_pos[1],h_pos[2]);
	((MultiNBodyWorld*)theWorld)->getMDB()->Buffer(h_pos,theWorld->getIter(),di);
}

__global__ void integrateBodies(float4* newPos,
                float4* oldPos,
                float4* vel,
                unsigned int deviceNumBodies,
                float deltaTime, float damping, float softeningSquared, int totalNumBodies)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    //printf("Index: %d\n",index); // Debug test for Kernel crash if not using SDK

    if (index >= totalNumBodies)
    {
        return;
    }

    float4 position = oldPos[index];

    float4 accel = computeBodyAccel(position, oldPos, totalNumBodies, softeningSquared);

    //Debug to ensure accel not NaN or 0
    //if(index==1)printf("Accel: %f %f %f at %f %f %f\n",accel.x,accel.y,accel.z,position.x,position.y,position.z);



    if (threadIdx.y == 0)
    {
        // acceleration = force \ mass;
        // new velocity = old velocity + acceleration * deltaTime
        // note we factor out the body's mass from the equation, here and in bodyBodyInteraction
        // (because they cancel out).  Thus here force == acceleration
        float4 velocity = vel[index];

        velocity.x += accel.x * deltaTime;
        velocity.y += accel.y * deltaTime;
        velocity.z += accel.z * deltaTime;

        velocity.x *= damping;
        velocity.y *= damping;
        velocity.z *= damping;

        // new position = old position + velocity * deltaTime
        position.x += velocity.x * deltaTime;
        position.y += velocity.y * deltaTime;
        position.z += velocity.z * deltaTime;

        // store new position and velocity
        newPos[index] = position;
        vel[index]    = velocity;
    }
}

void MultiNBodyDomain::gpuIntegrateNBodySystem(float deltaTime,int di)
{
    hipDeviceProp_t props;

    //unsigned int dev = 0;
    hipSetDevice(dev);
	hipGetDeviceProperties(&props, dev);

	// Tile Size = p*q
	// Tiles must be full
	// np%(pq)=0

	int q=1;
	int p=32;
	while(np%((2*q))==0)q*=2; // Np divisibility first


	q=min(q,props.multiProcessorCount);
	//int p=np/q;//props.maxThreadsPerBlock/(props.multiProcessorCount));

	q=min(props.multiProcessorCount,np/p);


	int sharedMemSize = p * q * 4 * sizeof(float); // 4 floats for pos
	while(sharedMemSize > props.sharedMemPerBlock/2){
            q/=2;
	    sharedMemSize = p * q * 4 * sizeof(float);
	}

	int g=(np + (p-1))/(p);

	dim3 threads(p,q,1);
	dim3 grid(g, 1, 1);

//// ----DEBUG---- Print Thread Block and Memory to Identify Some Kernel Crash Problems
////	printf("p,q=%d,%d x %d mem=%d / %d\n",p,q,g,sharedMemSize,props.sharedMemPerBlock);

	// execute the kernel:

	// When the numBodies / thread block size is < # multiprocessors
	// (16 on G80), the GPU is underutilized. For example, with 256 threads per
	// block and 1024 bodies, there will only be 4 thread blocks, so the
	// GPU will only be 25% utilized.  To improve this, we use multiple threads
	// per body.  We still can use blocks of 256 threads, but they are arranged
	// in q rows of p threads each.  Each thread processes 1/q of the forces
	// that affect each body, and then 1/q of the threads (those with
	// threadIdx.y==0) add up the partial sums from the other threads for that
	// body.  To enable this, use the "--p=" and "--q=" command line options to
	// this example.  e.g.: "nbody.exe --n=1024 --p=64 --q=4" will use 4
	// threads per body and 256 threads per block. There will be n/p = 16
	// blocks, so a G80 GPU will be 100% utilized.

	// We use a bool template parameter to specify when the number of threads
	// per body is greater than one, so that when it is not we don't have to
	// execute the more complex code required!

	int currentRead=theWorld->getIter()%2;

	//hipMemcpyAsync(h_pos,d_pos[currentRead],4*np*sizeof(float),hipMemcpyDeviceToHost,0);
	hipMemcpy(h_pos,d_pos[currentRead],4*np*sizeof(float),hipMemcpyDeviceToHost);

	integrateBodies<<< grid, threads, sharedMemSize >>>
			((float4*)d_pos[1-currentRead],(float4*)d_pos[currentRead],(float4*)d_mom, np,
					deltaTime, param.m_damping, param.m_softeningSquared, np);

#ifdef USE_SDK
	// check if kernel invocation generated an error
	getLastCudaError("Kernel execution failed");
#endif

	hipDeviceSynchronize();
	//printf("iter %d domain %d point 0 %f %f %f\n",theWorld->getIter(),di,h_pos[0],h_pos[1],h_pos[2]);	
	((MultiNBodyWorld*)theWorld)->getMDB()->Buffer(h_pos,theWorld->getIter(),di);
}

void MultiNBodyDomain::randomizeBodies(int config){
	if(onGpu())	hipMemset(d_force, 0, np*4*sizeof(float)); // Clear Force Vector

	switch (config)
	{
		default:
		case NBODY_CONFIG_RANDOM:
			{
				float scale = param.m_clusterScale * std::max<float>(1.0f, np / (1024.0f));
				float vscale = param.m_velocityScale * scale;

				int p = 0, v = 0;
				int i = 0;

				while (i < np)
				{
					float3 point;
					//const int scale = 16;
					point.x = rand() / (float) RAND_MAX * 2 - 1;
					point.y = rand() / (float) RAND_MAX * 2 - 1;
					point.z = rand() / (float) RAND_MAX * 2 - 1;
					float lenSqr = point.x*point.x+point.y*point.y+point.z*point.z;//dot(point, point);

					if (lenSqr > 1)
						continue;

					float3 velocity;
					velocity.x = rand() / (float) RAND_MAX * 2 - 1;
					velocity.y = rand() / (float) RAND_MAX * 2 - 1;
					velocity.z = rand() / (float) RAND_MAX * 2 - 1;
					lenSqr = velocity.x*velocity.x+velocity.y*velocity.y+velocity.z*velocity.z;//dot(velocity, velocity);

					if (lenSqr > 1)
						continue;

					h_pos[p++] = point.x * scale; // pos.x
					h_pos[p++] = point.y * scale; // pos.y
					h_pos[p++] = point.z * scale; // pos.z
					h_pos[p++] = 1.0f; // mass

					h_mom[v++] = h_pos[p]*velocity.x * vscale; // pos.x
					h_mom[v++] = h_pos[p]*velocity.y * vscale; // pos.x
					h_mom[v++] = h_pos[p]*velocity.z * vscale; // pos.x
					h_mom[v++] = (h_mom[v-1]*h_mom[v-1]+h_mom[v-2]*h_mom[v-2]+h_mom[v-3]*h_mom[v-3])/h_pos[p]; // energy

					i++;
				}
			}
			break;

		case NBODY_CONFIG_SHELL:
			{
				float scale = param.m_clusterScale;
				float vscale = scale * param.m_velocityScale;
				float inner = 2.5f * scale;
				float outer = 4.0f * scale;

				int p = 0, v=0;
				int i = 0;

				while (i < np)//for(int i=0; i < numBodies; i++)
				{
					float x, y, z;
					x = rand() / (float) RAND_MAX * 2 - 1;
					y = rand() / (float) RAND_MAX * 2 - 1;
					z = rand() / (float) RAND_MAX * 2 - 1;

					float3 point = {x, y, z};
					float len = sqrt(point.x*point.x+point.y*point.y+point.z*point.z);//normalize(point);
					point.x/=len;point.y/=len;point.z/=len;

					if (len > 1)
						continue;

					h_pos[p++] =  point.x * (inner + (outer - inner) * rand() / (float) RAND_MAX);
					h_pos[p++] =  point.y * (inner + (outer - inner) * rand() / (float) RAND_MAX);
					h_pos[p++] =  point.z * (inner + (outer - inner) * rand() / (float) RAND_MAX);
					h_pos[p++] = 1.0f;

					x = 0.0f; // * (rand() / (float) RAND_MAX * 2 - 1);
					y = 0.0f; // * (rand() / (float) RAND_MAX * 2 - 1);
					z = 1.0f; // * (rand() / (float) RAND_MAX * 2 - 1);

					float3 axis = {x, y, z};
					len=sqrt(axis.x*axis.x+axis.y*axis.y+axis.z*axis.z);//normalize(axis);
					axis.x/=len;axis.y/=len;axis.z/=len;

					if (1 - point.x*axis.x+point.y*axis.y+point.z*axis.z < 1e-6)  //dot(point, axis)
					{
						axis.x = point.y;
						axis.y = point.x;
						len=sqrt(axis.x*axis.x+axis.y*axis.y+axis.z*axis.z);//normalize(axis);
						axis.x/=len;axis.y/=len;axis.z/=len;
					}

					//if (point.y < 0) axis = scalevec(axis, -1);
					float3 vv = {(float)h_pos[4*i], (float)h_pos[4*i+1], (float)h_pos[4*i+2]};
					vv = make_float3(vv.y*axis.z-vv.z*axis.y,vv.z*axis.x-vv.x*axis.z,vv.x*axis.y-vv.y*axis.x);//cross(vv, axis);
					h_mom[v++] = h_pos[p] * vv.x * vscale;
					h_mom[v++] = h_pos[p] * vv.y * vscale;
					h_mom[v++] = h_pos[p] * vv.z * vscale;
					h_mom[v++] = (h_mom[v-1]*h_mom[v-1]+h_mom[v-2]*h_mom[v-2]+h_mom[v-3]*h_mom[v-3])/h_pos[p]; // energy

					i++;
				}
			}
			break;

		case NBODY_CONFIG_EXPAND:
			{
				float scale = param.m_clusterScale * np / (1024.f);

				if (scale < 1.0f)
					scale = param.m_clusterScale;

				float vscale = scale * param.m_velocityScale;

				int p = 0, v = 0;

				for (int i=0; i < np;)
				{
					float3 point;

					point.x = rand() / (float) RAND_MAX * 2 - 1;
					point.y = rand() / (float) RAND_MAX * 2 - 1;
					point.z = rand() / (float) RAND_MAX * 2 - 1;

					float lenSqr = point.x*point.x+point.y*point.y+point.z*point.z;//dot(point, point);

					if (lenSqr > 1)
						continue;

					h_pos[p++] = point.x * scale; // pos.x
					h_pos[p++] = point.y * scale; // pos.y
					h_pos[p++] = point.z * scale; // pos.z
					h_pos[p++] = 1.0f; // mass
					h_mom[v++] = h_pos[p] * point.x * vscale; // pos.x
					h_mom[v++] = h_pos[p] * point.y * vscale; // pos.x
					h_mom[v++] = h_pos[p] * point.z * vscale; // pos.x
					h_mom[v++] = (h_mom[v-1]*h_mom[v-1]+h_mom[v-2]*h_mom[v-2]+h_mom[v-3]*h_mom[v-3])/h_pos[p]; // energy

					i++;
				}
			}
			break;
	}

	if(onGpu()){
		hipSetDevice(dev);
		hipMemcpy(d_pos[0],h_pos,4*np*sizeof(float),hipMemcpyHostToDevice);
		hipMemcpy(d_mom,h_mom,4*np*sizeof(float),hipMemcpyHostToDevice);
	}
}
